#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

# define MAX_THREADS_PER_BLOCK 1024
# define BLOCK_SIZE 8

__device__ int mandelDevice(float c_re, float c_im, int maxIterations) {
    float z_re = c_re, z_im = c_im;
    int i;

#pragma unroll

    for(i = 0; i < maxIterations; ++i) {
        if(z_re * z_re + z_im * z_im > 4.f) {
            break;
        }
        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int* d_img, int resX, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;  
    float c_re = lowerX + tid_x * stepX;
    float c_im = lowerY + tid_y * stepY;

    d_img[tid_y * resX + tid_x] = mandelDevice(c_re, c_im, maxIterations);

}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    /* allocate memory on host and device */
    int *d_img;
    size_t size = resX * resY * sizeof(int);
    hipMalloc((void**)&d_img, size);

    /* setup the execution config */
    /* beware of the limitation of block size */
    dim3 threads_per_block(BLOCK_SIZE, BLOCK_SIZE);
    if (threads_per_block.x * threads_per_block.y > MAX_THREADS_PER_BLOCK) {
        printf("The thread number exceeds the maximum thread number per block.\n");
        exit(1);
    }
    dim3 number_of_blocks(resX / threads_per_block.x, resY / threads_per_block.y);
    mandelKernel<<<number_of_blocks, threads_per_block>>>(lowerX, lowerY, stepX, stepY, d_img, resX, maxIterations);

    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));

    /* read output from device */
    hipMemcpy(img, d_img, size, hipMemcpyDeviceToHost);

    /* free device memory */
    hipFree(d_img);
}
